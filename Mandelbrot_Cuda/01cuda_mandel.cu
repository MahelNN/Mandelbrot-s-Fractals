#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "gpuerrchk.h"

/* Bounds of the Mandelbrot set */
#define X_MIN -1.78
#define X_MAX 0.78
#define Y_MIN -0.961
#define Y_MAX 0.961
/* numbers of threads per block */
#define NTHREADS 32

typedef struct {

  int nb_rows, nb_columns; /* Dimensions */
  char * pixels; /* Linearized matrix of pixels */

} Image;

static void error_options () {

  fprintf (stderr, "Use : ./mandel [options]\n\n");
  fprintf (stderr, "Options \t Meaning \t\t Default val.\n\n");
  fprintf (stderr, "-n \t\t Nb iter. \t\t 100\n");
  fprintf (stderr, "-b \t\t Bounds \t\t -1.78 0.78 -0.961 0.961\n");
  fprintf (stderr, "-d \t\t Dimensions \t\t 1024 768\n");
  fprintf (stderr, "-f \t\t File \t\t Image/mandel_cuda.ppm\n");
  exit (1);
}

static void analyzis (int argc, char * * argv, int * nb_iter, double * x_min, double * x_max, double * y_min, double * y_max, int * width, int * height, char * * path) {

  const char * opt = "b:d:n:f:" ;
  int c ;

  /* Default values */
  * nb_iter = 100;
  * x_min = X_MIN;
  * x_max = X_MAX;
  * y_min = Y_MIN;
  * y_max = Y_MAX;
  * width = 1024;
  * height = 768;
  
  * path = "Image/mandel_cuda.ppm";

  /* Analysis of arguments */
  while ((c = getopt (argc, argv, opt)) != EOF) {
    
    switch (c) {
      
    case 'b':
      sscanf (optarg, "%lf", x_min);
      sscanf (argv [optind ++], "%lf", x_max);
      sscanf (argv [optind ++], "%lf", y_min);
      sscanf (argv [optind ++], "%lf", y_max);
      break ;
    case 'd': /* width */
      sscanf (optarg, "%d", width);
      sscanf (argv [optind ++], "%d", height);
      break;
    case 'n': /* Number of iterations */
      * nb_iter = atoi (optarg);
      break;
    case 'f': /* Output file */
      * path = optarg;
      break;
    default :
      error_options ();
    };
  }  
}

static void initialization (Image * im, int nb_columns, int nb_rows) {
  im -> nb_rows = nb_rows;
  im -> nb_columns = nb_columns;
  im -> pixels = (char *) malloc (sizeof (char) * nb_rows * nb_columns); /* Space memory allocation */
} 

static void save (const Image * im, const char * path) {
  /* Image saving using the ASCII format'.PPM' */
  unsigned i;
  FILE * f = fopen (path, "w");  
  fprintf (f, "P6\n%d %d\n255\n", im -> nb_columns, im -> nb_rows); 
  for (i = 0; i < im -> nb_columns * im -> nb_rows; i ++) {
    char c = im -> pixels [i];
    fprintf (f, "%c%c%c", c, 2*c, c); /* Monochrome weight */
  }
  fclose (f);
}


__global__ static void    
cuda_Compute (char * pixels, int nb_rows, int nb_columns, int nb_iter, double x_min, double x_max, double y_min, double y_max) {
  
  unsigned int l = blockIdx.y * blockDim.y + threadIdx.y; /* Global indice line */
  unsigned int c = blockIdx.x * blockDim.x + threadIdx.x; /* Global indice column */
  unsigned int pos;
  double dx, dy, a, b, x, y;
  int i = 0;

  dx = (x_max - x_min) / nb_columns;
  dy = (y_max - y_min) / nb_rows; /* Discretization */

  if(l<nb_rows || l>0)
    if(c<nb_columns || c>0)
      {
	
	pos = l * nb_columns + c; /* Position of the computed pixel */
	/* Computation at each point of the image */
	a = x_min + c * dx;
	b = y_max - l * dy;
	x = 0, y = 0;      

	i=0;
	while (i < nb_iter) {
	  double tmp = x;
	  x = x * x - y * y + a;
	  y = 2 * tmp * y + b;
	  if (x * x + y * y > 4) /* Divergence ! */
	    break; 
	  else
	    i++;
	}     
	pixels[ pos++ ] = (double) i / nb_iter * 255;
      } 
}


double Choose_Tserial(int N)
{
  double Tserial;
  switch (N)
    {
    case 100:
      Tserial = 0.08354276;
      break;      
    case 200:
      Tserial = 0.15676422;
      break;      
    case 250:
      Tserial = 0.19308734;
      break;
    case 500:
      Tserial = 0.37361187; 
      break;
    case 750:
      Tserial = 0.55358624;
      break;
    case 1000:
      Tserial = 0.73245299;
      break;
    default:
      Tserial = 0.08354276;
      break;
    }
  return Tserial;
}


int main (int argc, char * * argv)
{
    
  int nb_iter, width, height; /* Degree of precision, dimensions of the image */  
  double x_min, x_max, y_min, y_max; /* Bounds of representation */
  char * path; /* File destination */
  Image im;
  char * im_d_pixels;

  struct timespec t0, t1, t2, t3;
  double Tserial, Tparallel, Tcomm;
  FILE *fptr;

  analyzis(argc, argv, & nb_iter, & x_min, & x_max, & y_min, & y_max, & width, & height, & path);
  initialization (& im, width, height); /* Initialisation on the Host */

  /* Initialization on the Device */
  hipMalloc( & im_d_pixels, height * width * sizeof(char) );

  /* Mapping on the Device */
  dim3 gridDim ( ( width + NTHREADS - 1 ) / NTHREADS , ( height + NTHREADS-1 ) / NTHREADS ) ;
  dim3 blockDim ( NTHREADS, NTHREADS );

  /* Computation on the Device */
  clock_gettime(CLOCK_MONOTONIC, &t0);
  cuda_Compute <<< gridDim , blockDim >>> ( im_d_pixels, height, width, nb_iter, x_min, x_max, y_min, y_max );
  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &t1);
  
  /* Copy data from Device to Host */
  clock_gettime(CLOCK_MONOTONIC, &t2);
  gpuErrchk( hipMemcpy( im.pixels, im_d_pixels, height * width * sizeof( char ), hipMemcpyDeviceToHost ) );
  clock_gettime(CLOCK_MONOTONIC, &t3);

  /* save as a picture */
  save (& im, path);

  Tparallel = (t1.tv_sec-t0.tv_sec)+(t1.tv_nsec-t0.tv_nsec)/1e9f;
  Tcomm = (t3.tv_sec-t2.tv_sec)+(t3.tv_nsec-t2.tv_nsec)/1e9f;
   
  printf(" Tparallel = %2.9lf\n Tcomm = %2.9lf\n height=%d, width=%d \n", Tparallel, Tcomm, height, width);
      
  Tserial  = Choose_Tserial(nb_iter);  /* grisou-8 */
   
  fptr = fopen("Data/01speedup.dat","a"); 
  fprintf(fptr, "%2.9lf %2.9lf %2.9lf %2.9lf\n", Tserial, Tparallel+Tcomm, Tparallel, Tcomm);
  fclose(fptr);

  /* free the resources on the Device and Host */
  hipFree( im_d_pixels );
  free( im.pixels );
  return 0 ;
}
